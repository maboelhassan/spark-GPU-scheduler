#include <stdio.h>
#include <stdlib.h>
#include <netdb.h>
#include <netinet/in.h>
#include <string.h>
#include <unistd.h>
#include <pthread.h>
#include "cuda_kernels.cuh"

#define MESSAGE_LENGTH 10
#define MAX_MESSAGE_LENGTH 134217728
#define INPUT_MESSAGE_LENGTH 67108864
#define OUTPUT_MESSAGE_LENGTH 3145728
#define GPU_COUNT 1
#define GPU_STREAM_COUNT 4 
#define MAX_THREADS_COUNT 100
#define streams_count 4

struct request_info{
      int thread_number;
      int socket;
      int kafka_message_id;
      int log_id;
      hipStream_t * stream;
};

int recv_all(int sockfd, void *buffer, int length, int flags){
    int current_length = length;
    char  *buffer_ptr = (char*) buffer;
    int bytes_count = 0 ;
    while (current_length > 0){
        int bytes_read = recv(sockfd, buffer_ptr, current_length, flags);
        
        if(bytes_read <= 0){
              return bytes_read;
        }

        current_length -= bytes_read; 
        buffer_ptr += bytes_read;
        bytes_count += bytes_read;
    }
    return bytes_count;
}


int send_all(int sockfd, void *buffer, int length, int flags){
    int current_length = length;
    char *buffer_ptr = (char*)buffer;
    int bytes_count = 0;
    while(current_length > 0 ){
        int bytes_sent = send(sockfd, buffer_ptr, current_length, flags);
        
        if(bytes_count == 0){
            return bytes_count;
        }
        
        current_length -=bytes_sent;
        buffer_ptr += bytes_sent;
        bytes_count -= bytes_sent;
    }
    return bytes_count;
}




void * process(void * arg){
      short * buffer = (short *) malloc(INPUT_MESSAGE_LENGTH + 10);
      int check_operation ;
      struct request_info * curr_request_info = (struct request_info *) arg;
      int thread_number = curr_request_info->thread_number;
      int socket = curr_request_info->socket;
      int kafka_message_id = curr_request_info->kafka_message_id;
      int GPU_device = thread_number % GPU_COUNT ;

      hipStream_t * GPU_stream = curr_request_info->stream;
      int log_id = curr_request_info->log_id;
      
      // continue with the logging here
      
      printf("before accepting data");
      int bytes_count = recv_all(socket, buffer, INPUT_MESSAGE_LENGTH , 0);
      if(bytes_count <= 0){
            perror("error reading data from socket");
      }
      
      int message_length_shorts = INPUT_MESSAGE_LENGTH / 2 ;
      int results_count = 0 ;
      int output_message_doubles = OUTPUT_MESSAGE_LENGTH / 8 ;
      double results[output_message_doubles];
      int i;
      for(i =0 ; i< output_message_doubles; i++){
          results[i] = -1;
      }
	
      entry(buffer, results, &results_count, kafka_message_id, message_length_shorts, GPU_stream);
      send_all(socket, results, OUTPUT_MESSAGE_LENGTH, 0);
      free(buffer);
      pthread_exit(NULL);
}


int main( int argc, char *argv[] ) {
   int log_counter = 0 ;
   int thread_count = 0 ;
   int socket_file_desc, new_socket_file_desc, port_number;
   socklen_t clilen;
   char buffer[256];
   struct sockaddr_in serv_addr, cli_addr;
   
   
   socket_file_desc = socket(AF_INET, SOCK_STREAM, 0);
   if (socket_file_desc < 0) {      
      perror("error can't open socket_arg");
      exit(1);
   }

   bzero((char *) &serv_addr, sizeof(serv_addr));
   port_number = 5001;
   serv_addr.sin_family = AF_INET;
   serv_addr.sin_addr.s_addr = INADDR_ANY;
   serv_addr.sin_port = htons(port_number);


   if (bind(socket_file_desc, (struct sockaddr *) &serv_addr, sizeof(serv_addr)) < 0) {
      perror("erro can't bind");
      exit(1);
   }

   listen(socket_file_desc,15);
   clilen = sizeof(cli_addr);
   
   pthread_t threads[MAX_THREADS_COUNT];
   hipStream_t streams[streams_count];
   struct request_info req_info_arr[MAX_THREADS_COUNT];
   
   int i;
   for(i =0 ; i< streams_count; i++){
      hipError_t error = hipStreamCreate(&streams[i]);
      if(error != hipSuccess){
          printf("Error while creating stream %d error is %s",i,hipGetErrorString(error));
      }
   }

   while (1) {
      printf("before staring thread %d\n",thread_count);
      new_socket_file_desc = accept(socket_file_desc, (struct sockaddr *) &cli_addr, &clilen);
	
      printf("after starting thread %d\n",thread_count);	
      if (new_socket_file_desc < 0) {
         perror("error can't accept connections");
         exit(1);
      }
            
        int current_thread_number = thread_count ;
        req_info_arr[current_thread_number].thread_number = thread_count ;   
        req_info_arr[current_thread_number].socket = new_socket_file_desc ;
        /*
            kafka_message_id is always set to zero. needs to be handled later
        */
        req_info_arr[current_thread_number].kafka_message_id = 0;
        req_info_arr[current_thread_number].log_id = log_counter;
        req_info_arr[current_thread_number].stream = &streams[current_thread_number % streams_count];
        
        thread_count ++;
        log_counter  ++;
        thread_count = thread_count % MAX_THREADS_COUNT ;
        
        
	  int check_operation = pthread_create(&threads[current_thread_number], NULL, process, (void *)&req_info_arr[current_thread_number]);
        
	  if(check_operation){
		  printf("error couldn't create thread\n");
	  }
	  log_counter ++ ;
   }
}
